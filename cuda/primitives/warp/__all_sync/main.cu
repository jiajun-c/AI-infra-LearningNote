#include <hip/hip_runtime.h>
#include <iostream>
#define N 1000
__global__ void vote_all(int *a, int *b, int n)
 {
     int tid = threadIdx.x;
     if (tid > n)
         return;
     int temp = a[tid];
     b[tid] = __all_sync(0xffffffff, temp > 24); 
 }

int main() {
    int *a = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++) a[i] = i/24;
    int *b = (int *)malloc(N * sizeof(int));
    int* dev_a;
    int* dev_b;
    hipMalloc(&dev_a, N*sizeof(int));
    hipMalloc(&dev_b, N*sizeof(int));

    hipMemcpy(dev_a, a,  N*sizeof(int), hipMemcpyHostToDevice);
    vote_all<<<1, N>>>(dev_a, dev_b, N);
    hipDeviceSynchronize();
    hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d ", b[i]);
    }
    free(a);
    free(b);
    hipFree(dev_a);
    hipFree(dev_b);
}