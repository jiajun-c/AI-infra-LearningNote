#include <hip/hip_runtime.h>
#include <iostream>
#define N 1000
__global__ void ballot(int *a, int *b, int n)
 {
     int tid = threadIdx.x;
     if (tid > n)
         return;
     b[tid] = __ballot_sync(0xffffffff, a[tid] > 12);
 }

int main() {
    int *a = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++) a[i] = i%24;
    int *b = (int *)malloc(N * sizeof(int));
    int* dev_a;
    int* dev_b;
    hipMalloc(&dev_a, N*sizeof(int));
    hipMalloc(&dev_b, N*sizeof(int));

    hipMemcpy(dev_a, a,  N*sizeof(int), hipMemcpyHostToDevice);
    ballot<<<1, N>>>(dev_a, dev_b, N);
    hipDeviceSynchronize();
    hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d ", b[i]);
    }
    free(a);
    free(b);
    hipFree(dev_a);
    hipFree(dev_b);
}