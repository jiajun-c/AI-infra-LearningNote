#include <hip/hip_runtime.h>
#include <iostream>
#define N 1000
__global__ void shfl(int *a, int *b, int n)
 {
     int tid = threadIdx.x;
     int value = tid & 0x1f;
     if (tid > n)
         return;
     int temp = a[tid];
     value = __shfl_sync(0xffffffff, value, 2, 8);
     b[tid] = value;
 }

int main() {
    int *a = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++) a[i] = i%24;
    int *b = (int *)malloc(N * sizeof(int));
    int* dev_a;
    int* dev_b;
    hipMalloc(&dev_a, N*sizeof(int));
    hipMalloc(&dev_b, N*sizeof(int));

    hipMemcpy(dev_a, a,  N*sizeof(int), hipMemcpyHostToDevice);
    shfl<<<1, N>>>(dev_a, dev_b, N);
    hipDeviceSynchronize();
    hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d ", b[i]);
    }
    free(a);
    free(b);
    hipFree(dev_a);
    hipFree(dev_b);
}