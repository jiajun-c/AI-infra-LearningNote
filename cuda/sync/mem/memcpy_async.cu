#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <cassert>

// 简单的向量加法计算函数
__device__ void compute(int* global_out, const int* shared_in, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        global_out[idx] = shared_in[idx] + 1; // 简单的加1操作
    }
}

// 使用memcpy_async的内核函数
__global__ void with_memcpy_async(int* global_out, const int* global_in, size_t size, size_t batch_sz, 
                                  float* kernel_time_ms) {
    auto grid = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();
    if (block.size() > 0 && grid.block_index().x == 0 && block.group_index().x  ==0)
    printf(" grid.size %d  block.size %d\n", grid.size(), block.size());
  assert(size == batch_sz * grid.size()); // Exposition: input size fits batch_sz * grid_size

    extern __shared__ int shared[]; // 动态共享内存
    
    // 创建事件用于内核内部计时
    
    for (size_t batch = 0; batch < batch_sz; ++batch) {
        size_t block_batch_idx = block.group_index().x * block.size() + grid.size() * batch;
        
        // 整个线程组协作地将整个批次拷贝到共享内存
        cooperative_groups::memcpy_async(block, shared, global_in + block_batch_idx, 
                                       sizeof(int) * block.size());
        
        // 等待所有拷贝完成
        cooperative_groups::wait(block);
        
        compute(global_out + block_batch_idx, shared, block.size());
        
        block.sync();
    }
}

// 传统同步版本的内核函数（用于对比）
__global__ void traditional_sync(int* global_out, const int* global_in, size_t size, size_t batch_sz,
                                float* kernel_time_ms) {
    extern __shared__ int shared[];
    int tid = threadIdx.x;
    

    for (size_t batch = 0; batch < batch_sz; ++batch) {
        size_t block_batch_idx = blockIdx.x * blockDim.x + gridDim.x * blockDim.x * batch;
        size_t global_idx = block_batch_idx + tid;
        
        if (global_idx < size) {
            // 传统的同步拷贝：先到寄存器，再到共享内存
            int temp = global_in[global_idx];
            shared[tid] = temp;
        }
        
        __syncthreads(); // 等待所有拷贝完成
        
        compute(global_out + block_batch_idx, shared, blockDim.x);
        
        __syncthreads(); // 等待计算完成
    }

}

// 性能测试函数
void benchmark_memcpy_async(size_t total_size, size_t batch_size, int block_size) {
    // 分配主机内存（使用固定内存以获得最佳性能）
    int* h_input = nullptr;
    int* h_output_async = nullptr;
    int* h_output_traditional = nullptr;
    
    hipHostMalloc(&h_input, total_size * sizeof(int));
    hipHostMalloc(&h_output_async, total_size * sizeof(int));
    hipHostMalloc(&h_output_traditional, total_size * sizeof(int));
    
    // 初始化数据
    for (size_t i = 0; i < total_size; ++i) {
        h_input[i] = static_cast<int>(i);
    }
    
    // 分配设备内存
    int* d_input = nullptr;
    int* d_output_async = nullptr;
    int* d_output_traditional = nullptr;
    float* d_kernel_time_async = nullptr;
    float* d_kernel_time_traditional = nullptr;
    
    hipMalloc(&d_input, total_size * sizeof(int));
    hipMalloc(&d_output_async, total_size * sizeof(int));
    hipMalloc(&d_output_traditional, total_size * sizeof(int));
    hipMalloc(&d_kernel_time_async, sizeof(float));
    hipMalloc(&d_kernel_time_traditional, sizeof(float));
    
    // 计算网格和块尺寸
    size_t batch_sz = total_size / batch_size;
    dim3 block(block_size);
    dim3 grid((batch_size + block_size - 1) / block_size);
    
    size_t shared_mem_size = block_size * sizeof(int);
    hipMemcpy(d_input, h_input, total_size * sizeof(int), hipMemcpyHostToDevice);

    // 测试memcpy_async版本
    for (int i = 0; i < 20; ++i) {
        with_memcpy_async<<<grid, block, shared_mem_size>>>(d_output_async, d_input, total_size, batch_sz, d_kernel_time_async);
    }
    
    auto start_time = std::chrono::high_resolution_clock::now();

    with_memcpy_async<<<grid, block, shared_mem_size>>>(
        d_output_async, d_input, total_size, batch_sz, d_kernel_time_async);
    hipDeviceSynchronize();
    auto end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_async, d_output_async, total_size * sizeof(int), hipMemcpyDeviceToHost);
    
    auto async_duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    
    float async_kernel_time;
    hipMemcpy(&async_kernel_time, d_kernel_time_async, sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
  traditional_sync<<<grid, block, shared_mem_size>>>(
        d_output_traditional, d_input, total_size, batch_sz, d_kernel_time_traditional);    }
    // 测试传统同步版本
    start_time = std::chrono::high_resolution_clock::now();
    
    traditional_sync<<<grid, block, shared_mem_size>>>(
        d_output_traditional, d_input, total_size, batch_sz, d_kernel_time_traditional);
    hipDeviceSynchronize();
    end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_output_traditional, d_output_traditional, total_size * sizeof(int), hipMemcpyDeviceToHost);
    
    auto traditional_duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    
    float traditional_kernel_time;
    hipMemcpy(&traditional_kernel_time, d_kernel_time_traditional, sizeof(float), hipMemcpyDeviceToHost);
    
    // 验证结果正确性
    bool results_match = true;
    for (size_t i = 0; i < total_size; ++i) {
        if (h_output_async[i] != h_output_traditional[i]) {
            results_match = false;
            break;
        }
    }
    
    // 打印性能结果
    std::cout << "=== 性能测试结果 ===" << std::endl;
    std::cout << "数据大小: " << total_size << " 个整数 (" << total_size * sizeof(int) / 1024.0 << " KB)" << std::endl;
    std::cout << "批次大小: " << batch_size << ", 块大小: " << block_size << std::endl;
    std::cout << "批次数: " << batch_sz << std::endl;
    std::cout << std::endl;
    
    std::cout << "memcpy_async版本:" << std::endl;
    std::cout << "  - 总执行时间: " << async_duration.count() << " μs" << std::endl;
    // std::cout << "  - 内核执行时间: " << async_kernel_time << " ms" << std::endl;
    std::cout << "  - 总吞吐量: " << (total_size * sizeof(int) / 1024.0 / 1024.0) / (async_duration.count() / 1000000.0) << " MB/s" << std::endl;
    
    std::cout << "传统同步版本:" << std::endl;
    std::cout << "  - 总执行时间: " << traditional_duration.count() << " μs" << std::endl;
    // std::cout << "  - 内核执行时间: " << traditional_kernel_time << " ms" << std::endl;
    std::cout << "  - 总吞吐量: " << (total_size * sizeof(int) / 1024.0 / 1024.0) / (traditional_duration.count() / 1000000.0) << " MB/s" << std::endl;
    std::cout << std::endl;
    
    std::cout << "性能提升:" << std::endl;
    // std::cout << "  - 内核时间加速比: " << traditional_kernel_time / async_kernel_time << "x" << std::endl;
    std::cout << "  - 总时间加速比: " << traditional_duration.count() / (double)async_duration.count() << "x" << std::endl;
    std::cout << "  - 结果正确性: " << (results_match ? "✓ 通过" : "✗ 失败") << std::endl;
    
    // 清理资源
    hipHostFree(h_input);
    hipHostFree(h_output_async);
    hipHostFree(h_output_traditional);
    hipFree(d_input);
    hipFree(d_output_async);
    hipFree(d_output_traditional);
    hipFree(d_kernel_time_async);
    hipFree(d_kernel_time_traditional);
}
int main() {
    // 设置不同的测试场景
    std::vector<std::tuple<size_t, size_t, int>> test_cases = {
        {1024 * 1024, 4096, 256}   // 中等数据量，中等批次
        // {2048 * 2048, 8192, 512},    // 大数据量，大批次
        // {512 * 512, 2048, 128}   ,    // 小数据量，小批次
        // {512 * 32, 32, 32}       // 小数据量，小批次
    };
    
    std::cout << "开始memcpy_async性能测试..." << std::endl;
    std::cout << "==========================================" << std::endl;
    
    for (const auto& test_case : test_cases) {
        size_t total_size = std::get<0>(test_case);
        size_t batch_size = std::get<1>(test_case);
        int block_size = std::get<2>(test_case);
        
        benchmark_memcpy_async(total_size, batch_size, block_size);
        std::cout << "==========================================" << std::endl;
    }
    
    return 0;
}