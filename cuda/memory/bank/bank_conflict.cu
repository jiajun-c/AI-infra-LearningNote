#include <hip/hip_runtime.h>
#include <iostream>

__global__ void transSmem(float* out) {
    __shared__ float matrix[32][32];
    matrix[threadIdx.y][threadIdx.x ^ threadIdx.y] = out[threadIdx.x + 32*threadIdx.y]; // 加载原始数据
    __syncthreads();
    out[threadIdx.x + 32*threadIdx.y] = matrix[threadIdx.x][threadIdx.x ^ threadIdx.y]; // 写入转置数据
}

int main() {
    dim3 block{32, 32};
    float *d_out, *out;
    hipMalloc((void**)&d_out, 32*32*4);
    out = (float*)malloc(sizeof(float)*32*32);
    for (int i = 0; i < 32*32; i++) {
        out[i] = i;
    }
    hipMemcpy(d_out, out, 32*32*sizeof(float), hipMemcpyHostToDevice);
    transSmem<<<1, block>>>(d_out);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, 32*32*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 32;i++) {
        for (int j = 0; j < 32; j++) {
            printf("%.2f ", out[i*32+j]);
        }
        printf("\n");
    }

}